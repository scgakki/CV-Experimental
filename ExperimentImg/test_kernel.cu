#include "hip/hip_runtime.h"
#ifndef  __TEST_KERNEL_CU_
#define  __TEST_KERNEL_CU_
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <iostream>
#define datasize 100

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void add(int *c, int *a, int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

extern "C" void add_host(int *host_a, int *host_b, int *host_c)
{
	int *dev_a, *dev_b, *dev_c;
	checkCudaErrors(hipMalloc((void**)&dev_a, sizeof(int)* datasize));
	checkCudaErrors(hipMalloc((void**)&dev_b, sizeof(int)* datasize));
	checkCudaErrors(hipMalloc((void**)&dev_c, sizeof(int)* datasize));

	checkCudaErrors(hipMemcpy(dev_a, host_a, sizeof(int)* datasize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_b, host_b, sizeof(int)* datasize, hipMemcpyHostToDevice));

	add<< <1, datasize >> >(dev_c, dev_a, dev_b);

	checkCudaErrors(hipMemcpy(host_c, dev_c, sizeof(int)* datasize, hipMemcpyDeviceToHost));
	hipFree(dev_a);//�����Կ��ڴ�  
	hipFree(dev_b);
	hipFree(dev_c);
	
}

#endif // ! __TEST_KERNEL_CU_